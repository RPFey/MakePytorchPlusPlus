#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>

#include "src/common.hpp"
#include "src/utils.hpp"

template <typename Dtype>
__global__ void sum(Dtype *a, Dtype *b, Dtype *c, int N) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= N) {
    c[i] = a[i] + b[i];
  }
}

template <typename Dtype>
void AddGPU(at::Tensor in_a, at::Tensor in_b, at::Tensor out_c) {
  if (in_a.numel() != in_b.numel())
    throw std::invalid_argument(Formatter()
                                << "Size mismatch A.numel(): " << in_a.numel()
                                << ", B.numel(): " << in_b.numel());

  out_c.resize_({in_a.numel()});

  hipError_t err;

  sum<<<GET_BLOCKS(in_a.numel()), CUDA_NUM_THREADS, 0,
        at::cuda::getCurrentCUDAStream()>>>(in_a.data<Dtype>(),
                                            in_b.data<Dtype>(),
                                            out_c.data<Dtype>(), in_a.numel());

  err = hipGetLastError();
  if (hipSuccess != err)
    throw std::runtime_error(Formatter()
                             << "CUDA kernel failed : " << std::to_string(err));
}

template void AddGPU<float>(at::Tensor in_a, at::Tensor in_b, at::Tensor out_c);
